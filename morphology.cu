#include <hip/hip_runtime.h>

extern "C"
{
    __global__ void dilation(int * src, int * dst, int p, int window_size, int n_window, int image_shape)
    {
        extern __shared__ int smem[];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;

        if (tx == 0)
        {
            for (int i = p - 1; i >= 0; i--)
            {
                if (i == p - 1)
                    smem[ty * p + i] = src[bx * p * n_window + ty * p + i];
                else
                    smem[ty * p + i] = max(src[bx * p * n_window + ty * p + i], smem[ty * p + (i + 1)]);
            }
        }
        else
        {
            for (int i = 0; i <= p - 1; i++)
            {
                if (i == 0)
                    smem[n_window * p + (ty * p) + i] = src[bx * p * n_window + ty * p + (i + p - 1)];
                else
                    smem[n_window * p + (ty * p) + i] = max(src[bx * p *n_window + ty * p + (i + p - 1)],
                                                 smem[n_window * p + (ty * p) + (i - 1)]);
            }
        }
        __syncthreads();

        if (tx == 0)
        {
            for (int i = 0; i < p; i++)
            {
                // Skip first p-1 / 2 because of padding
                int original_index = bx * p * n_window + ty * p + i + ((p - 1)/2);

                if (original_index < image_shape)
                {
                    dst[original_index] = max(smem[ty * p + i], smem[n_window * p + (ty * p) + i]);
                }
            }
        }

    }
}

extern "C"
{
    __global__ void erosion(int * src, int * dst, int p, int window_size, int n_window, int image_shape)
    {
        extern __shared__ int smem[];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;

        if (tx == 0)
        {
            for (int i = p - 1; i >= 0; i--)
            {
                if (i == p - 1)
                    smem[ty * p + i] = src[bx * p * n_window + ty * p + i];
                else
                    smem[ty * p + i] = min(src[bx * p * n_window + ty * p + i], smem[ty * p + (i + 1)]);
            }
        }
        else
        {
            for (int i = 0; i <= p - 1; i++)
            {
                if (i == 0)
                    smem[n_window * p + (ty * p) + i] = src[bx * p * n_window + ty * p + (i + p - 1)];
                else
                    smem[n_window * p + (ty * p) + i] = min(src[bx * p *n_window + ty * p + (i + p - 1)],
                                                 smem[n_window * p + (ty * p) + (i - 1)]);
            }
        }
        __syncthreads();

        if (tx == 0)
        {
            for (int i = 0; i < p; i++)
            {
                int original_index = bx * p * n_window + ty * p + i + ((p - 1)/2);

                if (original_index < image_shape)
                {
                    dst[original_index] = min(smem[ty * p + i], smem[n_window * p + (ty * p) + i]);
                }
            }
        }

    }
}